

#include "hip/hip_runtime.h"


#include "stdio.h"
#include <iostream>

#define gpuErrChk(res) {gpuAssert((res), __FILE__, __LINE__);}

inline void gpuAssert(hipError_t ret, const char* file, int line, bool abort=true) {
  // check if ret was a success
  if (ret != hipSuccess) {
    std::cout << "gpu assert " << hipGetErrorString(ret) << file << " " << line << " \n";
    if (abort) exit(ret);
  }
}
